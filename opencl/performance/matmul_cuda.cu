#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

__global__ void Muld(double *, double *, double *, int); 

// USAGE: matmul <smaller> <larger> <step>
// OUTPUT: PERFORMANCE IN GFLOPS

#define CACHELINE 64
#define TRIALS 30
#define MAXTIME 5000000 /* 5 seconds */
#define BLOCK 32

int main ( int argc, char **argv ) {

	int i,j,n,p,nmin,nmax,step;
	double *a,*b,*c;
	struct timeval tstart,t1,t2;
	double ttstart,tt1,tt2,diff,min;

	if (argc < 4) return 0;
	nmin = atoi(argv[1]);
	nmax = atoi(argv[2]);
	step = atoi(argv[3]);
	if (nmin < 1 || nmin > nmax || step < 1) return 0;

/* allocate the three matrices and align to cache lines */
	a = (double *)malloc(nmax*nmax*sizeof(double)+CACHELINE);
	b = (double *)malloc(nmax*nmax*sizeof(double)+CACHELINE);
	c = (double *)malloc(nmax*nmax*sizeof(double)+CACHELINE);
	a = (double *)(((unsigned long)a+CACHELINE)&~(CACHELINE-1));
	b = (double *)(((unsigned long)b+CACHELINE)&~(CACHELINE-1));
	c = (double *)(((unsigned long)c+CACHELINE)&~(CACHELINE-1));

// Allocate 3 arrays on GPU
	double *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a,nmax*nmax*sizeof(double));
	hipMalloc((void **)&d_b,nmax*nmax*sizeof(double));
	hipMalloc((void **)&d_c,nmax*nmax*sizeof(double));


/* initialize A and B */
	for (i=0;i<nmax;i++)
		for (j=0;j<nmax;j++) {
			a[i*nmax+j] = j;
			b[i*nmax+j] = i;
		}

/* performance evaluation for all matrix size from nmin to nmax */
	for (n=nmin; n<=nmax; n += step) {

		for (;n % BLOCK; n++);

		min = 1.0e100;
		gettimeofday(&tstart,NULL);
		ttstart = (double)tstart.tv_sec*1.0e6 + (double)tstart.tv_usec;
/* for each matrix size, run at most TRIALS times */
		for (p=0; p<TRIALS; p++) {
			gettimeofday(&t1,NULL);

hipMemcpy(d_a,a,n*n*sizeof(double),hipMemcpyHostToDevice);
hipMemcpy(d_b,b,n*n*sizeof(double),hipMemcpyHostToDevice);
dim3 dimBlock(BLOCK, BLOCK); 
dim3 dimGrid(n / dimBlock.x, n / dimBlock.y); 
Muld<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,n); 
hipDeviceSynchronize();
hipMemcpy(c,d_c,n*n*sizeof(double),hipMemcpyDeviceToHost);

			gettimeofday(&t2,NULL);
			tt1 = (double)t1.tv_sec * 1.0e6 + (double)t1.tv_usec;
			tt2 = (double)t2.tv_sec * 1.0e6 + (double)t2.tv_usec;
			diff = tt2 - tt1;
/* take the best performance result */
			if (diff < min) min = diff;
/* ...at most TRIALS times no longer than MAXTIME */
			if ((tt2 - ttstart) > MAXTIME)
				break;
		}

/* print performance in GFLOPS */
		double dn = (double)n;
		fprintf(stderr,"%u %g\n",n,(2*dn*dn*dn-dn*dn)/(1000*min));

	}


	//Free GPU memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);	

	return 0;

}

__global__ void Muld(double *A, double *B, double *C, int n) 
{ 
    // Block index 
    int bx = blockIdx.x; 
    int by = blockIdx.y; 
 
    // Thread index 
    int tx = threadIdx.x; 
    int ty = threadIdx.y; 
 
    // Index of the first sub-matrix of A processed by the block 
    int aBegin = n * BLOCK * by; 
 
    // Index of the last sub-matrix of A processed by the block 
    int aEnd   = aBegin + n - 1; 
 
    // Step size used to iterate through the sub-matrices of A 
    int aStep  = BLOCK; 
 
    // Index of the first sub-matrix of B processed by the block 
    int bBegin = BLOCK * bx; 
 
    // Step size used to iterate through the sub-matrices of B 
    int bStep  = BLOCK * n; 
 
    // The element of the block sub-matrix that is computed 
    // by the thread 
    double Csub = 0; 
 
    // Loop over all the sub-matrices of A and B required to 
    // compute the block sub-matrix 
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) { 
 
        __shared__ double As[BLOCK][BLOCK]; 
        __shared__ double Bs[BLOCK][BLOCK]; 

        As[ty][tx] = A[a + n * ty + tx]; 
        Bs[ty][tx] = B[b + n * ty + tx]; 
        __syncthreads(); 
 
        for (int k = 0; k < BLOCK; ++k) 
            Csub += As[ty][k] * Bs[k][tx]; 
 
        __syncthreads(); 
    } 
 
    // Write the block sub-matrix to global memory; 
    // each thread writes one element 
    int c = n * BLOCK * by + BLOCK * bx; 
    C[c + n * ty + tx] = Csub; 
}
